#include "hip/hip_runtime.h"

// #include <cstdio>
#include <cstdio>
#include <cstdlib>
#include <unistd.h>
#include <cassert>
#include <time.h>
#include <cmath>
#include <sys/stat.h>
#include <openssl/sha.h>
#include <SFML/Graphics.hpp>
#include <SFML/Audio.hpp>



struct Config{
 
    int window_x;
    int window_y;
    int max_count;
    double R_2;
    double scale;
    double x_center;
    double y_center;
    double dx;
    double dy;
    int last_press;


    void print(){
        printf("window_x = %d\n"
               "window_y = %d\n"
               "R_2 = %lg\n"
               "Max count = %d\n"
               "scale= %lg\n"
               "dx = %lg\n"
               "dy = %lg\n"
               "x center = %lg\n"
               "y center = %lg\n", window_x, window_y, R_2, max_count, scale, dx, dy, x_center, y_center);
               fflush(stdout);
    }

    void init(int winx, int winy, int maxc, double _R_2, double _scale, double _x_center, double _y_center){
        this->window_x = winx;
        this->window_y = winy;
        this->max_count = maxc;
        this->R_2 = _R_2;
        this->scale = _scale;
        this->x_center = _x_center;
        this->y_center = _y_center;
        this->dx = 10 * _scale;
        this->dy = 10 * _scale;
        this->last_press = clock();
    }

    void update(){
        
        if (sf::Keyboard::isKeyPressed(sf::Keyboard::Right)){
            this->x_center += this->dx;
        }

        if (sf::Keyboard::isKeyPressed(sf::Keyboard::Left)){
            this->x_center -= this->dx;
        }

        if (sf::Keyboard::isKeyPressed(sf::Keyboard::Down)){  
            this->y_center += this->dy;
        }
        
        if (sf::Keyboard::isKeyPressed(sf::Keyboard::Up)){    
            this->y_center      -= this->dy;
        }
        
        if (sf::Keyboard::isKeyPressed(sf::Keyboard::Num0)){
            this->scale *= 1.05; 
            this->dx = 10*this->scale; 
            this->dy = 10*this->scale;
        }
        
        if (sf::Keyboard::isKeyPressed(sf::Keyboard::Num1)){
            this->scale /= 1.05f; 
            this->dx = 10*this->scale; 
            this->dy = 10*this->scale;
        }
        
        if (sf::Keyboard::isKeyPressed(sf::Keyboard::C)){    
            printf("x = %e\n y = %e\n scale = %e\n", this->x_center, this->y_center, this->scale);
        }
    }


};



__device__ const int pallete_size = 16;
__device__ const unsigned colors[pallete_size + 1] = {
        66    +   256 * 30    +   256 * 256 * 15    +   256 * 256 * 256 * 255,
        25    +   256 * 7     +   256 * 256 * 26    +   256 * 256 * 256 * 255,
        9     +   256 * 1     +   256 * 256 * 47    +   256 * 256 * 256 * 255,
        4     +   256 * 4     +   256 * 256 * 73    +   256 * 256 * 256 * 255,
        0     +   256 * 7     +   256 * 256 * 100   +   256 * 256 * 256 * 255,
        12    +   256 * 44    +   256 * 256 * 138   +   256 * 256 * 256 * 255,
        24    +   256 * 82    +   256 * 256 * 177   +   256 * 256 * 256 * 255,
        57    +   256 * 125   +   256 * 256 * 209   +   256 * 256 * 256 * 255,
        134   +   256 * 181   +   256 * 256 * 229   +   256 * 256 * 256 * 255,
        211   +   256 * 236   +   256 * 256 * 248   +   256 * 256 * 256 * 255,
        241   +   256 * 233   +   256 * 256 * 191   +   256 * 256 * 256 * 255,
        248   +   256 * 201   +   256 * 256 * 95    +   256 * 256 * 256 * 255,
        255   +   256 * 170   +   256 * 256 * 0     +   256 * 256 * 256 * 255,
        204   +   256 * 128   +   256 * 256 * 0     +   256 * 256 * 256 * 255,
        153   +   256 * 87    +   256 * 256 * 0     +   256 * 256 * 256 * 255,
        106   +   256 * 52    +   256 * 256 * 3     +   256 * 256 * 256 * 255,
         0    +       0       +           0         +   256 * 256 * 256 * 255,
};

__device__ unsigned get_color(int n, Config* cfg){
    if (n < cfg->max_count) {
        return colors[n%pallete_size];
    }else{
        return colors[pallete_size];
    }
}

__global__ void calculate_double(Config* cfg, unsigned* result){
    int curr = (blockIdx.x << 10) + threadIdx.x;
    int indexy = curr/cfg->window_x;
    int indexx = curr%cfg->window_x;
    
    double y0 =  ((double)indexy - (cfg->window_y >> 1))*cfg->scale + cfg->y_center;
    double x0 =  ((double)indexx - (cfg->window_x >> 1))*cfg->scale + cfg->x_center;
    double X = x0, 
           Y = y0;

    
    int N = 0;
    
    for (; N < cfg->max_count; N++){
      double x2 = X*X,
            y2 = Y*Y,
            xy = X*Y;
    
      double r2 = x2 + y2;
         
      if (r2 >= cfg->R_2) break;
          
      X = x2 - y2 + x0,
      Y = xy + xy + y0;
    }
    result[indexy * cfg->window_x + indexx]= get_color(N, cfg);

}

__global__ void calculate_float(Config* cfg, unsigned* result){
    int curr = (blockIdx.x << 10) + threadIdx.x;
    int indexy = curr/cfg->window_x;
    int indexx = curr%cfg->window_x;
    
    float y0 =  ((float)indexy - (cfg->window_y >> 1))*cfg->scale + cfg->y_center;
    float x0 =  ((float)indexx - (cfg->window_x >> 1))*cfg->scale + cfg->x_center;
    float X = x0, 
           Y = y0;

    
    int N = 0;
    
    for (; N < cfg->max_count; N++){
      float x2 = X*X,
            y2 = Y*Y,
            xy = X*Y;
    
      float r2 = x2 + y2;
         
      if (r2 >= cfg->R_2) break;
          
      X = x2 - y2 + x0,
      Y = xy + xy + y0;
    }
    result[indexy * cfg->window_x + indexx]= get_color(N, cfg);

}



void printf_fps(){
    static clock_t lastTime = clock();
    clock_t currentTime = clock();
    double fps = 1.f / ((double)(currentTime - lastTime)/ CLOCKS_PER_SEC);
    lastTime = currentTime;
    printf("%g\r", fps);
    fflush(stdout);
}


void render(Config* cfg){
    int prev_iterations = cfg->max_count;
    cfg->max_count = 8192;

    unsigned* field = NULL;     
    char* name = (char*)calloc(40, sizeof(int));
    sprintf(name, "images/%d.jpeg", clock());

    int pixels_count = cfg->window_x * cfg->window_y;
    int block_count = pixels_count / 1024 + 1;
    hipMallocManaged(&field, (block_count + 1) * 1024 * sizeof(unsigned));

    calculate_double<<<block_count, 1024>>>(cfg, field);
    hipDeviceSynchronize();

    sf::Image img;
    img.create(cfg->window_x, cfg->window_y, (sf::Uint8*)field);
    img.saveToFile(name);

    hipFree(field);
    free(name);
    cfg->max_count = prev_iterations;
}


void draw_mandelbrott(Config* cfg){

    sf::RenderWindow window(sf::VideoMode(cfg->window_x, cfg->window_y), "BLABLABLA");
    window.setPosition(sf::Vector2i(430, 200));

    sf::Image image;
    image.create(cfg->window_x, cfg->window_y, sf::Color::Cyan);

    sf::Texture texture;
    texture.loadFromImage(image);


    sf::Sprite sprite;
    sprite.setTexture(texture);


    unsigned* field = NULL;     

    int pixels_count = cfg->window_x * cfg->window_y;
    int block_count = pixels_count / 1024 + 1;

    hipMallocManaged(&field, (block_count + 1) * 1024 * sizeof(unsigned));


    while (window.isOpen()){
        if (sf::Keyboard::isKeyPressed(sf::Keyboard::Escape))     break;
        if ((clock()-cfg->last_press)/CLOCKS_PER_SEC >= 2 && sf::Keyboard::isKeyPressed(sf::Keyboard::P)){
            render(cfg);
            cfg->last_press = clock();
        }    
        cfg->update();

        calculate_float<<<block_count, 1024>>>(cfg, field);
        hipDeviceSynchronize();

       
        

        printf_fps();
        // printf("x = %4e y = %4e %4e\r",cfg->x_center, cfg->y_center, cfg->scale);
        // cfg->print();
        fflush(stdout);
        texture.update((sf::Uint8*)field, cfg->window_x, cfg->window_y, 0, 0);

        sprite.setTexture(texture);

        window.draw(sprite);
        window.display();
    }
    
    hipFree(field);
}



int main(){

   
    Config* cfg = NULL;
    hipMallocManaged(&cfg, sizeof(Config));
    cfg->init(1600, 900, 1024, 4,  1/120.f,  0,  0);
    
    
    draw_mandelbrott(cfg);
    
    cfg->init(7680, 4800, 8192, 4, 5.802513e-06        ,  -7.374954e-01,  -2.084319e-01);

    // render(cfg);


    hipFree(cfg);

    

  return 0;
}